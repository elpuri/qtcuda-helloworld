
#include <hip/hip_runtime.h>
__global__ void fillKernel(float* array) {
    array[threadIdx.x] = threadIdx.x * 0.5;
}

void fillGpuArray(float* array, int count) {
    fillKernel<<<1, count>>>(array);

}

